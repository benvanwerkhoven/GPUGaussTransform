#include "hip/hip_runtime.h"
#include <hipcub/hipcub.hpp>



#ifndef block_size_x
    #define block_size_x 128    //best for GTX 690
#endif

/*
 * This function performs the main body of work for computing the Gauss transform
 * The parallelization is such that one thread block is created
 * for each item in A, which is of size m. This implies that each thread block
 * does n (size of B) work.
 * The gradient computed in this function is reduced to a single value within the
 * thread block. The same is done for the cross term, which then needs to be
 * reduced in a second kernel. 
 */
template<typename T, int dim>
__device__ void GaussTransform_blocked_i(const T *A, const T *B,
                 int m, int n, T scale_sq, T *d_grad, T *d_cross_term) {

    int tx = threadIdx.x;

    // Specialize BlockReduce for a 1D block of block_size_x threads on type T
    typedef hipcub::BlockReduce<T, block_size_x> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    T cross_term = 0.0;
    T grad_i[dim];
    for (int d = 0; d < dim; d++) {
        grad_i[d] = 0.0;
    }

    //for (int i = 0; i < m; ++i) { //loop parallelized over threads blocks
    int i = blockIdx.x;
    if (i>=m) return;

    //loop parallelized over threads within thread block
    for (int j = tx; j<n; j+=block_size_x) {

        T dist_ij = 0;
        for (int d = 0; d < dim; ++d) {
            dist_ij += (A[i * dim + d] - B[j * dim + d])*(A[i * dim + d] - B[j * dim + d]);
        }
        T cost_ij = exp(-1.0 * dist_ij / scale_sq);

        for (int d = 0; d < dim; ++d) {
            grad_i[d] -= cost_ij * 2.0 * (A[i * dim + d] - B[j * dim + d]);
        }

        cross_term += cost_ij;
    }

    //reduce grad_i for each d, within the block (division by scale^2*m*n on CPU)
    for (int d = 0; d < dim; d++) {
        grad_i[d] = BlockReduce(temp_storage).Sum(grad_i[d]);
    }

    //reduce cross_term within the block, (division by m*n on CPU)
    cross_term = BlockReduce(temp_storage).Sum(cross_term);

    if (tx == 0 && blockIdx.x < m) {
        for (int d = 0; d < dim; d++) {
            d_grad[blockIdx.x * dim + d] = grad_i[d] / (scale_sq * m * n);
        }
        d_cross_term[blockIdx.x] = cross_term;
    }
}

extern "C"
__global__ void GaussTransform(const double* A, const double* B,
                 int m, int n, double scale_sq, double *grad, double *cross_term) {

    //2-dimensional with double precision
    GaussTransform_blocked_i<double, 2>(A, B, m, n, scale_sq, grad, cross_term);

}

/*
 * Reduce the per thread block cross terms computed in the GaussTransform kernel to single value
 * and divide by (m*n)
 *
 * This kernel is designed to run as single-thread block, because the number of terms to reduce is
 * of size n or m, which is expected to be around 2000 or so. The number of items to reduce
 * is passed as the last argument 'nblocks', which corresponds to the number of thread blocks used
 * by the first kernel.
 */
extern "C"
__global__ void reduce_cross_term(double *output, double *d_cross_term, int m, int n, int nblocks) {

    int tx = threadIdx.x;
    // Specialize BlockReduce for a 1D block of block_size_x threads on type T
    typedef hipcub::BlockReduce<double, block_size_x> BlockReduce;
    // Allocate shared memory for BlockReduce
    __shared__ typename BlockReduce::TempStorage temp_storage;

    double cross_term = 0.0;
    for (int i=tx; i<nblocks; i+=block_size_x) {
        cross_term += d_cross_term[i];
    }

    //reduce to single value within thread block
    cross_term = BlockReduce(temp_storage).Sum(cross_term);

    //thread 0 writes output
    if (tx == 0) {
        output[0] = cross_term / (m*n);
    }

}



/*
 * Host part for calling GPUGaussTransform from the CPU 
 */

class GPUGaussTransform {
    public:
        GPUGaussTransform(int max_n);
        ~GPUGaussTransform();
        double compute(const double *A, const double *B, int m, int n, double scale, double *grad);
    private:
        int dim;
        int max_n;
        double *d_A;
        double *d_B;
        double *d_grad;
        double *d_cross_term;

        hipStream_t stream;
};


GPUGaussTransform::GPUGaussTransform(int n) {
    //allocate GPU memory for size max_n
    max_n = n;
    dim = 2;
    int elems = max_n * dim;

    hipError_t err;

    err = hipMalloc((void **)&d_A, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_B, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_grad, elems*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMalloc((void **)&d_cross_term, max_n*sizeof(double));
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMalloc: %s\n", hipGetErrorString(err));
        exit(1);
    }

    err = hipStreamCreate(&stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipStreamCreate: %s\n", hipGetErrorString(err));
        exit(1);
    }

    hipDeviceSynchronize();
} 

GPUGaussTransform::~GPUGaussTransform() {
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_grad);
    hipFree(d_cross_term);
    hipStreamDestroy(stream);
} 

double GPUGaussTransform::compute(const double *A, const double *B,
    int m, int n, double scale, double *grad) {

    double energy;
    hipError_t err;

    //move data to the GPU
    err = hipMemcpyAsync(d_A, A, m*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync: %s\n", hipGetErrorString(err));
        exit(1);
    }
    err = hipMemcpyAsync(d_B, B, n*dim*sizeof(double), hipMemcpyHostToDevice, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyAsync: %s\n", hipGetErrorString(err));
        exit(1);
    }

    //setup kernel execution parameters
    dim3 threads(block_size_x, 1, 1);
    dim3 grid(m, 1, 1);
    
    //call the first kernel
    double scale_sq = scale * scale;
    GaussTransform<<<grid, threads, 0, stream>>>(d_A, d_B, m, n, scale_sq, d_grad, d_cross_term); 

    //call the second kernel
    dim3 grid2(1, 1, 1);
    reduce_cross_term<<<grid2, threads, 0, stream>>>(d_cross_term, d_cross_term, m, n, m);

    //copy result from GPU memory to host memory
    err = hipMemcpyAsync(grad, d_grad, m*dim*sizeof(double), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyDeviceToHost: %s\n", hipGetErrorString (err));
        exit(1);
    }

    err = hipMemcpyAsync(&energy, d_cross_term, 1*sizeof(double), hipMemcpyDeviceToHost, stream);
    if (err != hipSuccess) {
        fprintf(stderr, "Error in hipMemcpyDeviceToHost: %s\n", hipGetErrorString (err));
        exit(1);
    }

    return energy;
}



extern "C"
float test_GaussTransformHost(double *cost, const double* A, const double* B,
            int m, int n, int dim, double scale, double* grad) {

    GPUGaussTransform gpu_gt(m);

    *cost = gpu_gt.compute(A, B, m, n, scale, grad);

    return 0.0;
}


